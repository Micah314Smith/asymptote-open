#include "hip/hip_runtime.h"
﻿/**
* @file kernel.cu
* @author Supakorn "Jamie" Rassameemasmuang <jamievlin@outlook.com>
* CUDA Kernel for computing irradiance by solid angle integration
*/

#include "kernel.h"
#include "helper.cuh"
#include "linalg.cuh"

#include <hip/hip_runtime.h>
#include <texture_indirect_functions.h>
#include <>

// Can we encode this somewhere else?
__device__ constexpr int PHI_SAMPLES = 300;
__device__ constexpr int THETA_SAMPLES = 400;

__device__ constexpr float THETA_INTEGRATION_REGION = HALFPI;
__device__ constexpr float PHI_INTEGRATION_REGION = TAU;
__device__ constexpr float dx_int_scale = 
    (THETA_INTEGRATION_REGION * PHI_INTEGRATION_REGION) / (PHI_SAMPLES * THETA_SAMPLES);

// #define TEST_NO_INTEGRAL

__global__
void irradiate(hipTextureObject_t tObjin, float3* out, size_t width, size_t height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;


    if (idx < width && idx_y < height)
    {
        int access_idx = to_idx(width, idx, idx_y);

        out[access_idx] = make_float3(0, 0, 0);

        float target_phi = TAU * ((idx + 0.5f) / width);
        float target_theta = PI * ((idx_y + 0.5f) / height);

        float3 N = from_sphcoord(target_phi, target_theta);
        float3 N1 = make_float3(
            __cosf(target_theta) * __cosf(target_phi),
            __cosf(target_theta) * __sinf(target_phi),
            -1*__sinf(target_theta));
        float3 N2 = make_float3(-1 * __sinf(target_phi), __cosf(target_phi), 0);

#ifndef TEST_NO_INTEGRAL
        for (int i = 0; i < PHI_SAMPLES; ++i)
        {
            float sampled_phi = i * PHI_INTEGRATION_REGION / PHI_SAMPLES;
            for (int j = 0; j < THETA_SAMPLES; ++j)
            {
                float sampled_theta = j * THETA_INTEGRATION_REGION / THETA_SAMPLES;

                // vec3 is the world space coordinate
                float2 sphcoord = to_sphcoord(angleToBasis(N, N1, N2, sampled_phi, sampled_theta));

                float4 frag = tex2D<float4>(tObjin,
                    sphcoord.x * PI_RECR * width / 2,
                    sphcoord.y * PI_RECR * height);

                float3 frag3 = make_float3(frag.x, frag.y, frag.z);
                float scale = PI_RECR * dx_int_scale * __cosf(sampled_theta)* __sinf(sampled_theta);

                float3_addinplace(out[access_idx], frag3, scale);
            }
        }
#else
        float2 sphcoord = to_sphcoord(angleToBasis(N, N1, N2, 0, 0));

        float4 frag = tex2D<float4>(tObjin,
            sphcoord.x * PI_RECR * width / 2,
            sphcoord.y * PI_RECR * height);

        float3 frag3 = make_float3(frag.x, frag.y, frag.z);
        out[access_idx] = frag3;
#endif
    }
}

const size_t blkSz = 15;
void irradiate_ker(float4* in, float3* out, size_t width, size_t height)
{
    float4* d_ptr;
    size_t pitch;
    cudaErrorCheck(hipMallocPitch(
        &d_ptr, &pitch, width * sizeof(float4), height));
    cudaErrorCheck(hipMemcpy2D(d_ptr, pitch, in,
        width * sizeof(float4), width*sizeof(float4),
        height, hipMemcpyHostToDevice));

    hipResourceDesc cRD;
    memset(&cRD, 0, sizeof(hipResourceDesc));
    cRD.resType = hipResourceTypePitch2D;
    cRD.res.pitch2D.devPtr = d_ptr;
    cRD.res.pitch2D.width = width;
    cRD.res.pitch2D.height = height;
    cRD.res.pitch2D.desc = hipCreateChannelDesc<float4>();
    cRD.res.pitch2D.pitchInBytes = pitch;
    
    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.sRGB = 0;
    texDesc.readMode = hipReadModeElementType;

    hipTextureObject_t t_obj;
    cudaErrorCheck(hipCreateTextureObject(
        &t_obj, &cRD, &texDesc, nullptr));

    // out source
    float3* d_out;
    cudaErrorCheck(hipMalloc(
        (void**)&d_out, static_cast<size_t>(sizeof(float3) * width * height)));
    dim3 blockSz((width / blkSz) + 1, (height / blkSz) + 1);
    dim3 kerSz(blkSz, blkSz);
    irradiate KERNEL_ARGS(blockSz, kerSz) (t_obj, d_out, width, height);

    cudaErrorCheck(hipMemcpy(
        out, d_out, sizeof(float3) * width * height, hipMemcpyDeviceToHost));
    cudaErrorCheck(hipDestroyTextureObject(t_obj));
    hipFree(d_ptr);
}

